#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "tga.h"
#include "Stopwatch.h"
#include <limits>

constexpr float PI = 3.1415962f;
constexpr float E  = 2.718281828f;

constexpr size_t        ITERS           = 256;
constexpr float         THRESHOLD       = 6.f;
constexpr float         CTHRESHOLD      = 2.f;
constexpr float         ZOOM            = 2.3f;
constexpr float2        OFFSET          = { 1.7f, 1.2f };

struct mat4x4 {
    float data[16];
    __host__ __device__ 
        float& at(uint8_t x, uint8_t y);
};

__host__ __device__
float& mat4x4::at(uint8_t x, uint8_t y) {
    return data[x * 4 + y];
}

struct mat3x3 {
    float data[9];
    __host__ __device__
        float& at(uint8_t x, uint8_t y);
};

__host__ __device__
float& mat3x3::at(uint8_t x, uint8_t y) {
    return data[x * 3 + y];
}

__host__ __device__ 
byte operator""_b(unsigned long long val)
{
    return static_cast<byte>(val);
}

template<typename T>
__host__ __device__ 
 inline T square(T x) {
    return x * x;
}

template<typename T>
__host__ __device__
inline T inverse(T x) {
    return static_cast<T>(1) / x;
}

__host__ __device__ 
float2 squareComplex(float2 complex) {
    return { square(complex.x) - square(complex.y), 2.f * complex.x * complex.y };
}

__host__ __device__ 
float nextComplexAbs(float2 complex) {
    return square(complex.x) + square(complex.y);
}

#pragma warning(disable : 4838)
void fractalHost(int2 dimensions, color* output)
{
    #pragma omp parallel
    for (size_t i = 0; i < dimensions.x * dimensions.y; ++i)
    {
        size_t x = i / dimensions.y;
        size_t y = i % dimensions.y;

        output[y * dimensions.x + x] = color{ 0_b,  0_b,  0_b };

        float aspectRatio = (float)dimensions.y / (float)dimensions.x;
        float2 point {  (float)x / (float)dimensions.x, 
                        (float)y / (float)dimensions.y};

        point.x *= aspectRatio * ZOOM;
        point.y *= ZOOM;

        point.x -= OFFSET.x;
        point.y -= OFFSET.y;

        float2 z {0.f, 0.f};
        size_t iter = 0;

        for (; iter < ITERS; ++iter)
        {
            z = squareComplex(z);
            z.x += point.x;
            z.y += point.y;

            if (nextComplexAbs(z) > THRESHOLD)
                break;
        }

        if (nextComplexAbs(z) > CTHRESHOLD)
            output[y * dimensions.x + x] = color{ (byte)iter,  0_b,  0_b };
    }
}
#pragma warning(default : 4838)



__global__ void fractalKernel(int2 dimensions, color* output)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    output[y * dimensions.x + x] = color{ 0_b,  0_b,  0_b };

    float aspectRatio = (float)dimensions.y / (float)dimensions.x;
    float2 point{ (float)x / (float)dimensions.x, (float)y / (float)dimensions.y };

    point.x *= aspectRatio * ZOOM;
    point.y *= ZOOM;

    point.x -= OFFSET.x;
    point.y -= OFFSET.y;

    float2 z{ 0.f, 0.f };
    size_t iter = 0;

    for (; iter < ITERS; ++iter)
    {
        z = squareComplex(z);
        z.x += point.x;
        z.y += point.y;

        if (nextComplexAbs(z) > THRESHOLD)
            break;
    }

    if (nextComplexAbs(z) > CTHRESHOLD)
        output[y * dimensions.x + x] = color{ (byte)iter,  0_b,  0_b };
}

mat3x3 createGaussianKernel(float theta)
{
    mat3x3 kernel;
    for(int i = 0; i < 3; ++i)
        for (int j = 0; j < 3; ++j)
        {
            kernel.at(i, j) = inverse(2.f * PI * square(theta)) * 
                std::pow(E, (square(i) + square(j)) / (2.f * square(theta)));
        }

    return kernel;
}

__host__ __device__ bool isOutside(int2 point, int2 dimensions) {

    if (point.x > dimensions.x - 1 || point.x < 0)
        return true;

    if (point.y > dimensions.y - 1 || point.y < 0)
        return true;

    return false;
}

__global__ void convoluteKernel(mat3x3 kernel, int2 dimensions, color* input, color* output)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    color current {0.f,0.f,0.f};

    for(int i = -1; i < 2; ++i)
        for (int j = -1; j < 2; ++j)
        {
            int2 point{ x,y };
            point.x += i;
            point.y += j;

            if (!isOutside(point, dimensions))
            {
                current.r += input[point.x * dimensions.y + y].r * kernel.at(i + 1, j + 1);
                current.g += input[point.x * dimensions.y + y].g * kernel.at(i + 1, j + 1);
                current.b += input[point.x * dimensions.y + y].b * kernel.at(i + 1, j + 1);
            }
        }

    output[x * dimensions.y + y] = current;
}

int main()
{
    constexpr uint32_t x = 512, 
                       y = 512;

    int2 canvas{ x, y };
    color* data     = nullptr;
    color* blurred  = nullptr; 

    hipMallocManaged(&data, x * y * sizeof(color));
    hipMallocManaged(&blurred, x * y * sizeof(color));

    dim3 noThreads = { 16, 16 };
    dim3 noBlocks  = { x / noThreads.x, y / noThreads.y };

    fractalKernel<<<noBlocks, noThreads>>>(canvas, data);
    hipDeviceSynchronize();

    constexpr float THETA = 1.6f;

    convoluteKernel<<<noBlocks, noThreads>>>(createGaussianKernel(THETA), canvas, data, blurred);
    hipDeviceSynchronize();

    color* host_data = new color[x * y];

    hipError_t rc = hipMemcpy(host_data, blurred, x*y*sizeof(color), hipMemcpyKind::hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if (rc > 0)
    {
        fprintf(stderr, "Failed to copy memory from device!\n");
    }

    hipFree(data);

    //sw.start();
    //fractalHost(canvas, host_data);
    //sw.stop();

    //fprintf(stdout, "CPU time %I64u ms\n", sw.get_time<std::chrono::milliseconds>().count());

    tga fractal(host_data, {x,y});
    fractal.write("test.tga");

    delete[] host_data;
    return 0;
}